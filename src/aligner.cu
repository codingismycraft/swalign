#include "hip/hip_runtime.h"

#include "aligner.h"
#include "big_array.h"
#include <cstdlib>
#include <sstream>
#include <iostream>
#include <stdexcept>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <vector>
#include <new>
#include <iomanip>
#include <assert.h>


#define THREADS_PER_BLOCK 256

namespace {

__device__ __forceinline__ int max_of_three(int a, int b, int c) {

    if (a < 0) {
        a = 0; // Ensure non-negative values
    }

    if (b < 0) {
        b = 0; // Ensure non-negative values
    }

    if (c < 0) {
        c = 0; // Ensure non-negative values
    }


    int max = a;
    if (b > max) max = b;
    if (c > max) max = c;
    return max;
}


__device__  __host__ int get_flat_index(int row, int col, int cols) {
    if (row < 0 || col < 0 || cols <= 0) {
        return -1;
    }
    return row * cols + col;
}

__device__ __host__ int get_value(int* matrix, int row, int col, int cols) {
    const int flat_index = get_flat_index(row, col, cols);
    if (flat_index < 0) {
        return 0; // Return 0 for out-of-bounds indices
    }
    return matrix[flat_index];
}


__global__ void update_cell_in_diagonal(
    int* matrix,
    int d,
    int cols,
    int cells_count,
    const char* strA,
    const char* strB,
    int match_score,
    int mismatch_penalty,
    int gap_penalty)
{
    const int thread_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_index < cells_count) {
        const int row = thread_index + ((d >= cols) ? (d - cols + 1) : 0) + 1;
        const int col = d - row + 1;
        const int matrix_flat_index = get_flat_index(row, col, cols );

        const int upper_v = get_value(matrix, row - 1, col, cols) + gap_penalty;
        const int left_v = get_value(matrix, row, col - 1, cols) + gap_penalty;

        int diagonal_v = get_value(matrix, row - 1, col - 1, cols) ;

        if (strA[col-1] == strB[row-1]) {
            diagonal_v += match_score;
        } else {
            diagonal_v += mismatch_penalty;
        }

        const int score = max_of_three(upper_v, left_v, diagonal_v);
        matrix[matrix_flat_index] = score;
    }
}

std::string format_with_commas(long long value) {
    std::string num = std::to_string(value);
    int insertPosition = num.length() - 3;
    while (insertPosition > 0) {
        num.insert(insertPosition, ",");
        insertPosition -= 3;
    }
    return num;
}

}


// Implementation of LocalAlignment class.
Aligner::Aligner( const std::string& horizontal_seq, const std::string& vertical_seq,
        int match_score, int mismatch_penalty, int gap_penalty, size_t max_alignments):
            m_horizontal_seq(horizontal_seq),
            m_vertical_seq(vertical_seq),
            m_match_score(match_score),
            m_mismatch_penalty(mismatch_penalty),
            m_gap_penalty(gap_penalty),
            m_max_alignments(max_alignments),
            m_max_score(0),
            m_rows(vertical_seq.length()+1),
            m_cols(horizontal_seq.length()+1),
            m_matrix_size(long(m_rows) * m_cols* sizeof(int))
{

    if (m_matrix_size <= 0) {
        throw std::invalid_argument("value must be non-negative");
    }

    m_matrix = (int*)malloc(m_matrix_size);
    if (!m_matrix) {
        throw std::bad_alloc();
    }
    memset(m_matrix, 0, m_matrix_size);
    initializeMatrix();
}

Aligner::~Aligner() {
    if (m_matrix) {
        free(m_matrix);
        m_matrix = nullptr;
    }
}

int Aligner::count_anti_diagonal_cells(int anti_diagonal_index) {
    const int start_i = (anti_diagonal_index - (m_cols - 1) > 0) ? (anti_diagonal_index - (m_cols - 1)) : 0;
    const int end_i = (anti_diagonal_index < m_rows - 1) ? anti_diagonal_index : (m_rows - 1);
    const int count = end_i - start_i + 1;
    return (count > 0) ? count : 0;
}

void Aligner::initializeMatrix() {
    // Allocate device memory
    m_max_score = 0;

    char *d_horizontal_seq = nullptr;
    char *d_vertical_seq = nullptr;

    if (hipMalloc((void**)&d_horizontal_seq, m_cols + 1) != hipSuccess) {
        std::cerr << "Error allocating memory for horizontal sequence on GPU" << std::endl;
        exit(1);
    }


    if (hipMalloc((void**)&d_vertical_seq, m_rows + 1) != hipSuccess) {
        std::cerr << "Error allocating memory for vertical sequence on GPU" << std::endl;
        hipFree(d_horizontal_seq);
        exit(1);
    }

    // Copy strings to device
    if (hipMemcpy(d_horizontal_seq, m_horizontal_seq.c_str(), m_cols, hipMemcpyHostToDevice) != hipSuccess) {
        std::cerr << "Error copying horizontal sequence to GPU" << std::endl;
        hipFree(d_horizontal_seq);
        hipFree(d_vertical_seq);
        exit(1);
    }
    if (hipMemcpy(d_vertical_seq, m_vertical_seq.c_str(), m_rows, hipMemcpyHostToDevice) != hipSuccess) {
        std::cerr << "Error copying vertical sequence to GPU" << std::endl;
        hipFree(d_horizontal_seq);
        hipFree(d_vertical_seq);
        exit(1);
    }

    // Allocate memory on the GPU
    int* d_a = nullptr;

    if (hipMalloc((void**)&d_a, m_matrix_size) != hipSuccess) {
        std::cerr << "Error allocating memory for matrix on GPU" << std::endl;
        hipFree(d_horizontal_seq);
        hipFree(d_vertical_seq);
        exit(1);
    }


    if (hipMemcpy(d_a, m_matrix, m_matrix_size, hipMemcpyHostToDevice) != hipSuccess) {
        std::cerr << "Error copying matrix to GPU" << std::endl;
        hipFree(d_a);
        hipFree(d_horizontal_seq);
        hipFree(d_vertical_seq);
        exit(1);
    }

    for (int index = 1; index < m_cols + m_rows - 1;  ++index) {
        const int count = count_anti_diagonal_cells(index);
        const int numBlocks = (count + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        update_cell_in_diagonal<<<numBlocks, THREADS_PER_BLOCK>>>(
                d_a,
                index,
                m_cols,
                count,
                d_horizontal_seq,
                d_vertical_seq,
                m_match_score,
                m_mismatch_penalty,
                m_gap_penalty
        );
    }

    hipDeviceSynchronize();

    // Copy the matrix from GPU to CPU
    if (hipMemcpy(m_matrix, d_a, m_matrix_size, hipMemcpyDeviceToHost) != hipSuccess) {
        std::cerr << "Error copying matrix from GPU" << std::endl;
        hipFree(d_a);
        hipFree(d_horizontal_seq);
        hipFree(d_vertical_seq);
        exit(1);
    }

    // Free the GPU memory
    hipFree(d_a);
    hipFree(d_horizontal_seq);
    hipFree(d_vertical_seq);

    findMaxScores();
     for (const auto& pos : m_max_positions) {
        traceback(pos.first, pos.second, "", "", "");
    }

}

void Aligner::print_matrix() const {
    for (int row = 0; row < m_rows; row++) {
        for (int col = 0; col < m_cols; col++) {
            printf("%3d ", getScore(row, col));
        }
        printf("\n");
    }
}

void Aligner::findMaxScores() {
    int current_max = -1;
    m_max_positions.clear();

    for (int row = 0; row < m_rows; row++) {
        for (int col = 0; col < m_cols; col++) {
            const int value = m_matrix[row * m_cols + col];

            if (value > current_max) {
                m_max_positions.clear();
            }
            if (value >= current_max) {
                current_max = value;
                m_max_positions.push_back(std::make_pair(row, col));
            }
        }
    }

    m_max_score = current_max;
}

const std::string& Aligner::getHorizontalSeq() const{
    return m_horizontal_seq;
}


const std::string& Aligner::getVerticalSeq() const {
    return m_vertical_seq;
}


int Aligner::getRowCount() const {
    return m_rows;
}

int Aligner::getColCount() const {
    return m_cols;
}

int Aligner::getScore(int row, int col) const {
    if (row < 0 || row >= m_rows || col < 0 || col >= m_cols) {
        throw std::out_of_range("Row or column index out of bounds: " + std::to_string(row) + ", " + std::to_string(col));
    }
    const int index = row * m_cols + col;
    return m_matrix[index];
}

const std::vector<std::string>& Aligner::getLocalAlignments() const {
    return m_local_alignments;
}


size_t Aligner::getNumberOfAlignments() const {
    return m_local_alignments.size();
}

int Aligner::getMaxScore() const {
    return m_max_score;
}


std::string Aligner::toString() const {
    std::ostringstream oss;
    const int width = 6;
    if (m_matrix == nullptr) return "";

    // Build horizontal separator
    std::string separator;
    // The separator needs to cover one extra for the row label column.
    for (int col = 0; col < getColCount() + 1; ++col) {
        separator += std::string(width, '_');
        if (col + 1 < getColCount() + 1) separator += "|";
    }
    separator += "\n";

    // Print top separator
    oss << separator;

    // Print header row: empty cell, then m_horizontal_seq characters as column headers
    oss << std::setw(width) << ' ';

    oss << "|" << std::setw(width) << ' ';
    for (int col = 0; col < getColCount()-1; ++col) {
        oss << "|" << std::setw(width) << m_horizontal_seq[col];
    }
    oss << "\n" << separator;

    // Print all matrix rows
    for (int row = 0; row < getRowCount(); ++row) {
        // First column: empty for first row, else m_vertical_seq character
        if (row == 0) {
            oss << std::setw(width) << ' ';
        } else {
            oss << std::setw(width) << m_vertical_seq[row - 1];
        }
        // Print all matrix columns for this row
        for (int col = 0; col < getColCount(); ++col) {
            oss << "|" << std::setw(width) << getScore(row, col);
        }
        oss << "\n" << separator;
    }

    return oss.str();
}

void Aligner::traceback(int row, int col, std::string x1, std::string x2, std::string a) {
    while(row >0 && col >0 && getScore(row, col) > 0) {
        if (m_local_alignments.size() >= m_max_alignments)
        {
            return;
        }

        const int row_coming_in = row;
        const int col_coming_in = col;

        const std::string a_coming_in = a;
        const std::string x1_coming_in = x1;
        const std::string x2_coming_in = x2;

        const int current_score = getScore(row, col);
        const int diagonal_row = row - 1;
        const int diagonal_col = col - 1;

        const bool valid_diagonal = (diagonal_row >= 0 && diagonal_col >= 0);
        const bool valid_up_col = (col - 1 >= 0);
        const bool valid_left_row = (row - 1 >= 0);

        bool already_moved = false;

        const bool same_char = (m_horizontal_seq[col-1] == m_vertical_seq[row-1]);

        if (same_char && valid_diagonal && getScore(diagonal_row, diagonal_col) + m_match_score == current_score) {
            if (!already_moved) {
                a = '*' + a_coming_in;
                x1 = m_horizontal_seq[col_coming_in - 1] + x1_coming_in;
                x2 = m_vertical_seq[row_coming_in  - 1] + x2_coming_in;
                row = diagonal_row;
                col = diagonal_col;
                already_moved = true;
            }
            else {
                assert (false && "Traceback logic error: already_moved should not be true here");
            }
        }

        if (!same_char && valid_diagonal && getScore(diagonal_row, diagonal_col) + m_mismatch_penalty == current_score) {
            if (!already_moved) {
                a = '|' + a_coming_in;
                x1 = m_horizontal_seq[col_coming_in - 1] + x1_coming_in;
                x2 = m_vertical_seq[row_coming_in  - 1] + x2_coming_in;
                row = diagonal_row;
                col = diagonal_col;
                already_moved = true;
            }
            else {
                if (m_local_alignments.size() < m_max_alignments) {
                    traceback(
                        diagonal_row,
                        diagonal_col,
                        m_horizontal_seq[col_coming_in - 1] + x1_coming_in,
                        m_vertical_seq[row_coming_in  - 1] + x2_coming_in,
                        '|' + a_coming_in
                   );
               }
            }
        }


        if (valid_left_row && getScore(row_coming_in  - 1, col_coming_in) + m_gap_penalty == current_score) {
            if (!already_moved) {
                a = ' ' + a;
                x1 = '_' + x1;
                x2= m_vertical_seq[row_coming_in  - 1] + x2;
                row -= 1;
                already_moved = true;
            }
            else {
                if (m_local_alignments.size() < m_max_alignments) {
                    traceback(
                        row_coming_in  - 1,
                        col_coming_in,
                        '_' + x1_coming_in,
                        m_vertical_seq[row_coming_in  - 1] + x2_coming_in,
                        ' ' + a_coming_in
                   );
                }
            }
        }

        if (valid_up_col && getScore(row_coming_in, col_coming_in  - 1) + m_gap_penalty == current_score) {
            if (!already_moved) {
                a = ' ' + a;
                x1 = m_horizontal_seq[col - 1] + x1;
                x2 = '_' + x2;
                col  -= 1;
                already_moved = true;
            }
            else {
                if (m_local_alignments.size() < m_max_alignments) {
                    traceback(
                        row_coming_in,
                        col_coming_in  - 1,
                        m_horizontal_seq[col_coming_in - 1] + x1_coming_in,
                        '_' + x2_coming_in,
                        ' ' + a_coming_in
                   );
                }
            }
        }

       assert(already_moved && "Traceback logic error");
       assert (row < row_coming_in || col < col_coming_in);
    }

    a = a + " " + std::to_string(evaluateScore(a));

    m_local_alignments.push_back("\n"  + x2 + "\n" + a + "\n" + x1 + "\n");
}

int32_t Aligner::evaluateScore(const std::string& alighmentStr) const {
    int starCount = 0;
    int pipeCount = 0;
    int spaceCount = 0;

    for (char ch : alighmentStr) {
        if (ch == '*') {
            ++starCount;
        } else if (ch == '|') {
            ++pipeCount;
        } else if (ch == ' ') {
            ++spaceCount;
        }
    }

    return starCount * m_match_score +
           pipeCount * m_mismatch_penalty +
           spaceCount * m_gap_penalty;
}




std::ostream& operator<<(std::ostream& os, const Aligner& obj) {
    os << "\n***************************************" << std::endl;

    int counter = 1;
    os << "\n--------------------------" << std::endl;
    for (const auto& aln : obj.getLocalAlignments()) {
         os << "Alignment num: " << counter++ << "\n";
         os << aln << std::endl;
         os << "--------------------------" << std::endl;
    }

    const auto x = format_with_commas(obj.getRowCount() * obj.getColCount());

    std::ostringstream oss;
    oss << "Matrix Size: " << obj.getRowCount() << " X " << obj.getColCount() << " = " << x << " elements" << std::endl;
    std::string output = oss.str();
    std::cout << output << std::endl;

    os << "\nNumber of Alignments ..: " << obj.getNumberOfAlignments()<< std::endl;
    os << "Max score .............: " << obj.getMaxScore()<< std::endl;
    return os;
}

