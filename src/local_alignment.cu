#include "hip/hip_runtime.h"

#include "local_alignment.h"

#include <sstream>
#include <iostream>
#include <stdexcept>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <vector>
#include <new>
#include <iomanip>
#include <assert.h>


#define THREADS_PER_BLOCK 256

__device__ __forceinline__ int max_of_three(int a, int b, int c) {

    if (a < 0) {
        a = 0; // Ensure non-negative values
    }

    if (b < 0) {
        b = 0; // Ensure non-negative values
    }

    if (c < 0) {
        c = 0; // Ensure non-negative values
    }


    int max = a;
    if (b > max) max = b;
    if (c > max) max = c;
    return max;
}


__device__  __host__ int get_flat_index(int row, int col, int cols) {
    if (row < 0 || col < 0 || cols <= 0) {
        return -1;
    }
    return row * cols + col;
}

__device__ __host__ int get_value(int* matrix, int row, int col, int cols) {
    const int flat_index = get_flat_index(row, col, cols);
    if (flat_index < 0) {
        return 0; // Return 0 for out-of-bounds indices
    }
    return matrix[flat_index];
}


__global__ void update_cell_in_diagonal(
    int* matrix,
    int d,
    int cols,
    int cells_count,
    const char* strA,
    const char* strB,
    int match_score,
    int mismatch_penalty,
    int gap_penalty)
{
    const int thread_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_index < cells_count) {
        const int row = thread_index + ((d >= cols) ? (d - cols + 1) : 0) + 1;
        const int col = d - row + 1;
        const int matrix_flat_index = get_flat_index(row, col, cols );

        const int upper_v = get_value(matrix, row - 1, col, cols) + gap_penalty;
        const int left_v = get_value(matrix, row, col - 1, cols) + gap_penalty;

        int diagonal_v = get_value(matrix, row - 1, col - 1, cols) ;

        if (strA[col-1] == strB[row-1]) {
            diagonal_v += match_score;
        } else {
            diagonal_v += mismatch_penalty;
        }

        const int score = max_of_three(upper_v, left_v, diagonal_v);
        matrix[matrix_flat_index] = score;
    }
}



// Implementation of LocalAlignment class.
LocalAlignmentFinder::LocalAlignmentFinder( const std::string& s1, const std::string& s2,
        int match_score, int mismatch_penalty, int gap_penalty, size_t max_alignments):
            m_sequence1(s1),
            m_sequence2(s2),
            m_match_score(match_score),
            m_mismatch_penalty(mismatch_penalty),
            m_gap_penalty(gap_penalty),
            m_max_alignments(max_alignments),
            m_max_score(0),
            m_rows(s2.length()+1),
            m_cols(s1.length()+1),
            m_matrix_size(long(m_rows) * m_cols* sizeof(int))
{
    std::cout << "LocalAlignmentFinder initialized with sequences of lengths: "
              << m_rows << " and " << m_matrix_size << std::endl;
    if (m_matrix_size <= 0) {
        throw std::invalid_argument("value must be non-negative");
    }

    m_matrix = (int*)malloc(m_matrix_size);
    if (!m_matrix) {
        throw std::bad_alloc();
    }
    memset(m_matrix, 0, m_matrix_size);
    initializeMatrix();
}

LocalAlignmentFinder::~LocalAlignmentFinder() {
    if (m_matrix) {
        free(m_matrix);
        m_matrix = nullptr;
    }
}

int LocalAlignmentFinder::count_anti_diagonal_cells(int anti_diagonal_index) {
    const int start_i = (anti_diagonal_index - (m_cols - 1) > 0) ? (anti_diagonal_index - (m_cols - 1)) : 0;
    const int end_i = (anti_diagonal_index < m_rows - 1) ? anti_diagonal_index : (m_rows - 1);
    const int count = end_i - start_i + 1;
    return (count > 0) ? count : 0;
}

void LocalAlignmentFinder::initializeMatrix() {
    // Allocate device memory
    char *d_strA, *d_strB;
    hipMalloc((void**)&d_strA, m_rows + 1);
    hipMalloc((void**)&d_strB, m_cols + 1);

    // Copy strings to device
    hipMemcpy(d_strA, m_sequence1.c_str(), m_rows + 1, hipMemcpyHostToDevice);
    hipMemcpy(d_strB, m_sequence2.c_str(), m_cols + 1, hipMemcpyHostToDevice);

    // Allocate memory on the GPU
    int* d_a;

    if (hipMalloc((void**)&d_a, m_matrix_size) != hipSuccess) {
        throw std::bad_alloc();
    }

    if (hipMemcpy(d_a, m_matrix, m_matrix_size, hipMemcpyHostToDevice) != hipSuccess) {
        hipFree(d_a);
        throw std::bad_alloc();
    }

    for (int index = 1; index < m_cols + m_rows - 1;  index++) {
        const int count = count_anti_diagonal_cells(index);
        const int numBlocks = (count + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        update_cell_in_diagonal<<<numBlocks, THREADS_PER_BLOCK>>>(
                d_a,
                index,
                m_cols,
                count,
                d_strA,
                d_strB,
                m_match_score,
                m_mismatch_penalty,
                m_gap_penalty
        );
    }

    hipDeviceSynchronize();

    // Copy the matrix from GPU to CPU
    if (hipMemcpy(m_matrix, d_a, m_matrix_size, hipMemcpyDeviceToHost) != hipSuccess) {
        hipFree(d_a);
        throw std::bad_alloc();
    }

    // Free the GPU memory
    hipFree(d_a);
    hipFree(d_strA);
    hipFree(d_strB);

    findMaxScores();
     for (const auto& pos : m_max_positions) {
        traceback(pos.first, pos.second, "", "", "");
    }

}

void LocalAlignmentFinder::print_matrix() const {
    for (int row = 0; row < m_rows; row++) {
        for (int col = 0; col < m_cols; col++) {
            printf("%3d ", getScore(row, col));
        }
        printf("\n");
    }
}

void LocalAlignmentFinder::findMaxScores() {
    int current_max = -1;
    m_max_positions.clear();

    for (int row = 0; row < m_rows; row++) {
        for (int col = 0; col < m_cols; col++) {
            const int value = m_matrix[row * m_cols + col];
            if (value > current_max) {
                m_max_positions.clear();
            }
            if (value >= current_max) {
                current_max = value;
                m_max_positions.push_back(std::make_pair(row, col));
            }
        }
    }
}

const std::string& LocalAlignmentFinder::getSequence1() const{
    return m_sequence1;
}


const std::string& LocalAlignmentFinder::getSequence2() const {
    return m_sequence2;
}


int LocalAlignmentFinder::getRowCount() const {
    return m_rows;
}

int LocalAlignmentFinder::getColCount() const {
    return m_cols;
}

int LocalAlignmentFinder::getScore(int row, int col) const {
    if (row < 0 || row >= m_rows || col < 0 || col >= m_cols) {
        throw std::out_of_range("Row or column index out of bounds");
    }
    const int index = row * m_cols + col;
    return m_matrix[index];
}

const std::vector<std::string>& LocalAlignmentFinder::getLocalAlignments() const {
    return m_local_alignments;
}

std::string LocalAlignmentFinder::toString() const {
    std::ostringstream oss;
    const int width = 6;
    if (m_matrix == nullptr) return "";

    // Build horizontal separator
    std::string separator;
    // The separator needs to cover one extra for the row label column.
    for (int col = 0; col < getColCount() + 1; ++col) {
        separator += std::string(width, '_');
        if (col + 1 < getColCount() + 1) separator += "|";
    }
    separator += "\n";

    // Print top separator
    oss << separator;

    // Print header row: empty cell, then m_sequence1 characters as column headers
    oss << std::setw(width) << ' ';

    oss << "|" << std::setw(width) << ' ';
    for (int col = 0; col < getColCount()-1; ++col) {
        oss << "|" << std::setw(width) << m_sequence1[col];
    }
    oss << "\n" << separator;

    // Print all matrix rows
    for (int row = 0; row < getRowCount(); ++row) {
        // First column: empty for first row, else m_sequence2 character
        if (row == 0) {
            oss << std::setw(width) << ' ';
        } else {
            oss << std::setw(width) << m_sequence2[row - 1];
        }
        // Print all matrix columns for this row
        for (int col = 0; col < getRowCount(); ++col) {
            oss << "|" << std::setw(width) << getScore(row, col);
        }
        oss << "\n" << separator;
    }

    return oss.str();
}

void LocalAlignmentFinder::traceback(int row, int col, std::string x1, std::string x2, std::string a) {
    while(row >0 && col >0 && getScore(row, col) > 0) {
        if (m_local_alignments.size() >= m_max_alignments)
        {
            return;
        }

        const int row_coming_in = row;
        const int col_coming_in = col;

        const std::string a_coming_in = a;
        const std::string x1_coming_in = x1;
        const std::string x2_coming_in = x2;

        const int current_score = getScore(row, col);
        const int diagonal_row = row - 1;
        const int diagonal_col = col - 1;

        const bool valid_diagonal = (diagonal_row >= 0 && diagonal_col >= 0);
        const bool valid_up_col = (col - 1 >= 0);
        const bool valid_left_row = (row - 1 >= 0);

        bool already_moved = false;


        if (valid_diagonal && getScore(diagonal_row, diagonal_col) + m_match_score == current_score) {
            if (!already_moved) {
                a = '*' + a_coming_in;
                x1 = m_sequence1[col_coming_in - 1] + x1_coming_in;
                x2 = m_sequence2[row_coming_in  - 1] + x2_coming_in;
                row = diagonal_row;
                col = diagonal_col;
                already_moved = true;
            }
            else {
                assert (false && "Traceback logic error: already_moved should not be true here");
            }
        }

        if (valid_diagonal && getScore(diagonal_row, diagonal_col) + m_mismatch_penalty == current_score) {
            if (!already_moved) {
                a = '|' + a_coming_in;
                x1 = m_sequence1[col_coming_in - 1] + x1_coming_in;
                x2 = m_sequence2[row_coming_in  - 1] + x2_coming_in;
                row = diagonal_row;
                col = diagonal_col;
                already_moved = true;
            }
            else {
                if (m_local_alignments.size() < m_max_alignments) {
                    traceback(
                        diagonal_row,
                        diagonal_col,
                        m_sequence1[col_coming_in - 1] + x1_coming_in,
                        m_sequence2[row_coming_in  - 1] + x2_coming_in,
                        '|' + a_coming_in
                   );
               }
            }
        }


        if (valid_left_row && getScore(row_coming_in  - 1, col_coming_in) + m_gap_penalty == current_score) {
            if (!already_moved) {
                a = ' ' + a;
                x1 = '_' + x1;
                x2= m_sequence2[row_coming_in  - 1] + x2;
                row -= 1;
                already_moved = true;
            }
            else {
                if (m_local_alignments.size() < m_max_alignments) {
                    traceback(
                        row_coming_in  - 1,
                        col_coming_in,
                        '_' + x1_coming_in,
                        m_sequence2[row_coming_in  - 1] + x2_coming_in,
                        ' ' + a_coming_in
                   );
                }
            }
        }

        if (valid_up_col && getScore(row_coming_in, col_coming_in  - 1) + m_gap_penalty == current_score) {
            if (!already_moved) {
                a = ' ' + a;
                x1 = m_sequence1[col - 1] + x1;
                x2 = '_' + x2;
                col  -= 1;
                already_moved = true;
            }
            else {
                if (m_local_alignments.size() < m_max_alignments) {
                    traceback(
                        row_coming_in,
                        col_coming_in  - 1,
                        m_sequence1[col_coming_in - 1] + x1_coming_in,
                        '_' + x2_coming_in,
                        ' ' + a_coming_in
                   );
                }
            }
        }

       assert(already_moved && "Traceback logic error");
       assert (row < row_coming_in || col < col_coming_in);
    }

    m_local_alignments.push_back("\n"  + x2 + "\n" + a + "\n" + x1 + "\n");
}



std::ostream& operator<<(std::ostream& os, const LocalAlignmentFinder& obj) {
    os << "\n***************************************" << std::endl;
    os << "Seq1: " << obj.getSequence1() << std::endl;
    os << "Seq2: " << obj.getSequence2() << std::endl;

    //os << "\nNumber of Alignments ..: " << obj.getNumberOfAlignments()<< std::endl;
    //os << "Max score .............: " << obj.getMaxScore()<< std::endl;

    int counter = 1;
    os << "\n--------------------------" << std::endl;
    for (const auto& aln : obj.getLocalAlignments()) {
         os << "Alignment num: " << counter++ << "\n";
         os << aln << std::endl;
         os << "--------------------------" << std::endl;
    }
    return os;
}

